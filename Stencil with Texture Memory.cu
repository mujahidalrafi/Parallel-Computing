#include "hip/hip_runtime.h"
﻿#include "stdio.h"
#include <hip/hip_runtime.h>

const int blockCount = 60;
const int threadsPerBlock = 256;
const int radius = 3;
const int arraySize = blockCount * threadsPerBlock;
const int arraySizeWithHalos = arraySize + 2 * radius;

texture<int, hipTextureType1D, hipReadModeElementType> texRef;

__global__ void Stencil_1d(int* out)
{
	__shared__ int temp[threadsPerBlock + 2 * radius];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x + radius;
	int lindex = threadIdx.x + radius;

	temp[lindex] = tex1D(texRef, gindex);
	if (threadIdx.x < radius)
	{
		temp[lindex - radius] = tex1D(texRef, gindex - radius); 
		temp[lindex + threadsPerBlock] = tex1D(texRef, gindex + threadsPerBlock);
	}

	// Synchronize (ensure all the data is available)
	__syncthreads();

	// Apply the stencil
	int result = 0;
	for (int offset = -radius; offset <= radius; offset++)
		result += temp[lindex + offset];

	// Store the result
	out[gindex - radius] = result;
}

int main()
{
	int h_in[arraySizeWithHalos]; // add halos in the main input array too for simplicity in the kernel code
	int h_out[arraySize];
	for (int i = 0; i < arraySize; i++)
	{
		h_in[radius + i] = i + 1;
		h_out[i] = 0;
	}

	for (int i = 0; i < radius; i++)
	{
		h_in[i] = h_in[arraySizeWithHalos - i - 1] = 0;
	}

	int* d_out;
	hipMalloc(&d_out, arraySize * sizeof(int));
	hipMemcpy(d_out, h_out, arraySize * sizeof(int), hipMemcpyHostToDevice);

	// 	Bind the device input array to the texture reference
	hipArray_t d_in;
	hipChannelFormatDesc channel = hipCreateChannelDesc<int>();
	hipMallocArray(&d_in, &channel, arraySizeWithHalos, 1, hipArrayDefault);
	hipMemcpyToArray(d_in, 0, 0, h_in, arraySizeWithHalos * sizeof(int), hipMemcpyHostToDevice);
	hipBindTextureToArray(texRef, d_in, channel);

	Stencil_1d <<<blockCount, threadsPerBlock >>> (d_out);

	hipMemcpy(h_out, d_out, arraySize * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_out);

	printf("Radius: %d\n", radius);
	printf("Input:\t\t|\t\tOutput:\n");
	for (int i = 0; i < arraySize; i++)
		printf("%d\t\t|\t\t%d\n", h_in[i + radius], h_out[i]);

	return 0;
}
