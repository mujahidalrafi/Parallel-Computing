﻿#include "stdio.h"
#include <hip/hip_runtime.h>

const int blockCount = 60;
const int threadsPerBlock = 256;
const int radius = 3;
const int arraySize = blockCount * threadsPerBlock;

__global__ void Stencil_1d(const int* in, int* out)
{
	__shared__ int temp[threadsPerBlock + 2 * radius];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + radius;

	// Read input elements into shared memory
	temp[lindex] = in[gindex];
	if (threadIdx.x < radius)
	{
		temp[lindex - radius] = in[gindex - radius];
		temp[lindex + threadsPerBlock] = in[gindex + threadsPerBlock];
	}

	// Synchronize (ensure all the data is available)
	__syncthreads();
	
	// Apply the stencil
	int result = 0;
	for (int offset = -radius; offset <= radius; offset++)
		result += temp[lindex + offset];

	// Store the result
	out[gindex] = result;
}

int main()
{
	const int arraySizeWithHalos = arraySize + 2 * radius;
	int h_in[arraySizeWithHalos]; // add halos in the main input array too for simplicity in the kernel code
	int h_out[arraySize];
	for (int i = 0; i < arraySize; i++)
	{
		h_in[radius + i] = i + 1;
		h_out[i] = 0;
	}

	for (int i = 0; i < radius; i++)
		h_in[i] = h_in[arraySizeWithHalos - i - 1]= 0;

	int *d_in, *d_out;
	hipMalloc(&d_in, arraySizeWithHalos * sizeof(int));
	hipMalloc(&d_out, arraySize * sizeof(int));

	hipMemcpy(d_in, h_in, arraySizeWithHalos * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_out, h_out, arraySize * sizeof(int), hipMemcpyHostToDevice);

	Stencil_1d<<<blockCount, threadsPerBlock >>>(d_in + radius, d_out);
	
	hipMemcpy(h_in, d_in, arraySizeWithHalos * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_out, d_out, arraySize * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out);

	printf("Radius: %d\n", radius);
	printf("Input:\t\t|\t\tOutput:\n");
	for (int i = 0; i < arraySize; i++)
		printf("%d\t\t|\t\t%d\n", h_in[i + radius], h_out[i]);

	return 0;
}

