#include "hip/hip_runtime.h"
﻿#include "stdio.h"
#include <hip/hip_runtime.h>

const int blockCount = 60;
const int threadsPerBlock = 256;
const int radius = 3;
const int arraySize = blockCount * threadsPerBlock;
const int arraySizeWithHalos = arraySize + 2 * radius;

__constant__ int d_in_c[arraySizeWithHalos];

__global__ void Stencil_1d(int* out)
{
	__shared__ int temp[threadsPerBlock + 2 * radius];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x + radius;
	int lindex = threadIdx.x + radius;
	// Read input elements into shared memory
	temp[lindex] = d_in_c[gindex];
	if (threadIdx.x < radius)
	{
		temp[lindex - radius] = d_in_c[gindex - radius];
		temp[lindex + threadsPerBlock] = d_in_c[gindex + threadsPerBlock];
	}

	// Synchronize (ensure all the data is available)
	__syncthreads();

	// Apply the stencil
	int result = 0;
	for (int offset = -radius; offset <= radius; offset++)
		result += temp[lindex + offset];

	// Store the result
	out[gindex - radius] = result;
}

int main()
{
	int h_in[arraySizeWithHalos]; // add halos in the main input array too for simplicity in the kernel code
	int h_out[arraySize];
	for (int i = 0; i < arraySize; i++)
	{
		h_in[radius + i] = i + 1;
		h_out[i] = 0;
	}

	for (int i = 0; i < radius; i++)
		h_in[i] = h_in[arraySizeWithHalos - i - 1] = 0;

	int * d_out;
	hipMalloc(&d_out, arraySize * sizeof(int));

	hipMemcpyToSymbol(HIP_SYMBOL(d_in_c), h_in, arraySizeWithHalos * sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpy(d_out, h_out, arraySize * sizeof(int), hipMemcpyHostToDevice);

	Stencil_1d << <blockCount, threadsPerBlock >> > (d_out);

	hipMemcpy(h_in, d_in_c, arraySizeWithHalos * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_out, d_out, arraySize * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_out);

	printf("Radius: %d\n", radius);
	printf("Input:\t\t|\t\tOutput:\n");
	for (int i = 0; i < arraySize; i++)
		printf("%d\t\t|\t\t%d\n", h_in[i + radius], h_out[i]);

	return 0;
}
